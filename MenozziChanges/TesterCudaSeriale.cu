#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
/* Default data type is double, default size is 1024. */
#include "lu.h"

/* Array initialization. */
static void init_array(int n,
                       DATA_TYPE POLYBENCH_2D(A, N, N, n, n))
{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      A[i][j] = ((DATA_TYPE)(i + 1) * (j + 1)) / n;
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int n,
                        DATA_TYPE POLYBENCH_2D(A, N, N, n, n))

{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
    {
      fprintf(stderr, DATA_PRINTF_MODIFIER, A[i][j]);
      if ((i * n + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
/*
#pragma omp declare target
static void kernel_lu(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n))
{
    int i, j, k;
    #pragma omp target data map(tofrom: A[0:n][0:n])
    {
        #pragma omp target teams num_teams(_PB_N/NTHREADS_GPU) thread_limit(NTHREADS_GPU)
        {
            for (k = 0; k < _PB_N; k++) {
              
                #pragma omp distribute parallel for simd num_threads(NTHREADS_GPU) schedule(static,NTHREADS_GPU)
                for (j = k + 1; j < _PB_N; j++) {
                    A[k][j] = A[k][j] / A[k][k];
                }

                #pragma omp distribute parallel for simd  num_threads(NTHREADS_GPU) schedule(dynamic,NTHREADS_GPU) 
                for (i = k + 1; i < _PB_N; i++) {
                    for (j = k + 1; j < _PB_N; j++) {
                        A[i][j] -= A[i][k] * A[k][j];
                    } 
                }
            }
        }
    }
}
#pragma omp end declare target
*/
/*
static void kernel_lu(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n))
{
    int i, j, k;
     #pragma omp target data map(tofrom: A[0:n][0:n])
    {
            
            for (k = 0; k < _PB_N; k++) {
                #pragma omp  parallel for simd num_threads(NTHREADS_GPU) schedule(static,NTHREADS_GPU)
                for (j = k + 1; j < _PB_N; j++) {
                    A[k][j] = A[k][j] /A[k][k];;
                }

                #pragma omp  parallel for simd  num_threads(NTHREADS_GPU) schedule(dynamic,NTHREADS_GPU) 
                for (i = k + 1; i < _PB_N; i++) {
                    for (j = k + 1; j < _PB_N; j++) {
                        A[i][j] -=  A[i][k] * A[k][j];
                    } 
                }
            }
      }
}
*/
/*
static void kernel_lu(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n))
{
    int i, j, k;
            for (k = 0; k < _PB_N; k++) {
               
                for (j = k + 1; j < _PB_N; j++) {
                    A[k][j] = A[k][j] / A[k][k];
                }
                for (i = k + 1; i < _PB_N; i++) {
                    for (j = k + 1; j < _PB_N; j++) {
                        A[i][j] -= A[i][k] * A[k][j];
                    } 
                }
            }
}
*/
/* CUDA kernel per il calcolo del ciclo su `j` */

__global__ void lu_division(DATA_TYPE *A, int n, int k) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (j > k && j < n) {
        A[k * n + j] = A[k * n + j] / A[k * n + k]; 
    }
}

/* CUDA kernel per il calcolo del ciclo su `i` e `j` */
__global__ void lu_elimination(DATA_TYPE *A, int n, int k) {
    int i = blockIdx.x + k + 1; // elemento sotto alla diagonale di inizio
    int j = threadIdx.x + blockIdx.y * blockDim.y;

    if (i < n && j > k && j < n) {
        A[i * n + j] -= A[i * n + k] * A[k * n + j];
    }
}

static void kernel_lu(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n)) {
    DATA_TYPE *d_A;
    size_t size = n * n * sizeof(DATA_TYPE);

    /* Allocazione memoria sulla GPU */
    hipMalloc((void **)&d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(128); // valore statico da controllare

    for (int k = 0; k < n; k++) {
        /* Calcolo del ciclo su `j` */
        dim3 blocksPerGrid((n - k + threadsPerBlock.x - 1) / threadsPerBlock.x); //-1 per utilizzare meglio la gestione delle matrici in griglia
        lu_division<<<blocksPerGrid, threadsPerBlock>>>(d_A, n, k);

        /* Sincronizzazione per evitare dipendenze */
        hipDeviceSynchronize(); //Differenza tra __syncthread() è che questo sincronizza tutta la gpu, l'altro blocco per blocco

        /* Calcolo del ciclo su `i` e `j` */
        dim3 threadsPerBlock2D(16, 16);// valore statico da cambiare
        dim3 blocksPerGrid2D((n - k + threadsPerBlock2D.x - 1) / threadsPerBlock2D.x,
                             (n - k + threadsPerBlock2D.y - 1) / threadsPerBlock2D.y);
        lu_elimination<<<blocksPerGrid2D, threadsPerBlock2D>>>(d_A, n, k);

        /* Sincronizzazione */
        hipDeviceSynchronize();
    }
 
    /* Copia i risultati dalla GPU alla CPU */
    hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

    /* Libera la memoria sulla GPU */
    hipFree(d_A);
}
void kernel_lu_serial(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n)) {
    for (int k = 0; k < n; k++) {
        for (int j = k + 1; j < n; j++) {
            A[k][j] = A[k][j] / A[k][k];
        }
        for (int i = k + 1; i < n; i++) {
            for (int j = k + 1; j < n; j++) {
                A[i][j] -= A[i][k] * A[k][j];
            }
        }
    }
}

void test_correctness(int n, DATA_TYPE POLYBENCH_2D(A_serial, N, N, n, n), DATA_TYPE POLYBENCH_2D(A_cuda, N, N, n, n)) {
    printf("Confronto risultati:\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (fabs(A_serial[i][j] - A_cuda[i][j]) > 1) {
                printf("Differenza trovata in A[%d][%d]: serial=%.6f, cuda=%.6f\n", i, j, A_serial[i][j], A_cuda[i][j]);
                return;
            }
        }
    }
    printf("I risultati sono equivalenti.\n");
}


int main(int argc, char **argv) {
    /* Retrieve problem size. */
    int n = N;

    /* Variable declaration/allocation. */
    POLYBENCH_2D_ARRAY_DECL(A_cuda, DATA_TYPE, N, N, n, n);

    /* Initialize array(s). */
    init_array(n, POLYBENCH_ARRAY(A_cuda));

    /* Start timer. */
    polybench_start_instruments;

    /* Run kernel. */
    kernel_lu(n, POLYBENCH_ARRAY(A_cuda));

    /* Stop and print timer. */
    polybench_stop_instruments;
    polybench_print_instruments;

    POLYBENCH_2D_ARRAY_DECL(A_serial, DATA_TYPE, N, N, n, n);
    init_array(n, POLYBENCH_ARRAY(A_serial));
    kernel_lu_serial(n, POLYBENCH_ARRAY(A_serial));

    /* Prevent dead-code elimination. All live-out data must be printed
       by the function call in argument. */
    polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(A_serial)));
    polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(A_cuda)));

    test_correctness(n, POLYBENCH_ARRAY(A_serial), POLYBENCH_ARRAY(A_cuda));

    /* Be clean. */
    POLYBENCH_FREE_ARRAY(A_serial);
    POLYBENCH_FREE_ARRAY(A_cuda);


    return 0;
}
